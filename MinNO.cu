#include "hip/hip_runtime.h"
#if defined(_WIN32)
	#include <conio.h>
	#include <windows.h>
	extern "C"{
		#include "getopt.h"
	}
	#include <direct.h>
	#define GetCurrentDir _getcwd
	#define sleep(x) Sleep(1000 * (x))
	#define FILE_SEPARATOR "\\"
#else
	#include <unistd.h>
	#define GetCurrentDir getcwd
	#define FILE_SEPARATOR "/"
#endif

#include <string>
#include <iostream>
#include <sstream>
#include <dirent.h>
#include <sys/types.h>
#include <fstream>

#include <sys/stat.h>

#include "Connection.h"

#include "include/minknow_api/acquisition.grpc.pb.h"
#include "include/minknow_api/analysis_configuration.grpc.pb.h"
#include "include/minknow_api/data.grpc.pb.h"
#include "include/minknow_api/device.grpc.pb.h"
#include "include/minknow_api/instance.grpc.pb.h"
#include "include/minknow_api/keystore.grpc.pb.h"
#include "include/minknow_api/log.grpc.pb.h"
#include "include/minknow_api/manager.grpc.pb.h"
#include "include/minknow_api/minion_device.grpc.pb.h"
#include "include/minknow_api/promethion_device.grpc.pb.h"
#include "include/minknow_api/protocol.grpc.pb.h"
#include "include/minknow_api/statistics.grpc.pb.h"

using namespace minknow_api::acquisition;
using namespace minknow_api::analysis_configuration;
using namespace minknow_api::data;
using namespace minknow_api::device;
using namespace minknow_api::instance;
using namespace minknow_api::keystore;
using namespace minknow_api::log;
using namespace minknow_api::manager;
using namespace minknow_api::minion_device;
using namespace minknow_api::promethion_device;
using namespace minknow_api::protocol;
using namespace minknow_api::statistics;

int main(int argc, char** argv) {
	
	std::string host = "localhost"; // Default minknow host
	int port = 8000; // Default minknow port
	
	int wait_time = 60;
	
	int verbose = 0;
	int help = 0;
	
	char c;
	
	while( ( c = getopt (argc, argv, "P:H:w:vh") ) != -1 ) {
		switch(c) {		
			case 'P':
				if(optarg) port = atoi(optarg);
				break;
			case 'H':
				if(optarg) host = optarg;
				break;
			case 'w':
				if(optarg) wait_time = atoi(optarg);
				break;
			case 'v':
				verbose = 1;
				break;
			case 'h':
				help = 1;
				break;	
			default:
				/* You won't actually get here. */
				break;
		}
	}
	
	int num_args = argc - optind;
	if (help || num_args != 2){
		std::cerr << "Usage: " << argv[0] << " [options] <amount_of_data> <fastq_directory>" << std::endl
			<< "Client that keeps track of the amount of data read in by a MinKNOW device from a given directory and stops the run once the required amount has been reached." << std::endl
			<< "amount_of_data is the amount of data you want MinKNOW to read in before ending the run." << std::endl
			<< "fastq_directory is the directory containing the fastq files that will be monitored to see how much data has been read in. This should be the default directory that MinKNOW writes to." << std::endl
			<< "Options are:" << std::endl
			<< "[-H Host to open a connection on] default=" << host << std::endl
			<< "[-P Port to connect to] default=" << port << std::endl
			<< "[-w Wait time between directory checks. Measured in seconds] default=" << wait_time << std::endl
			<< "[-v verbose mode]" << std::endl
			<< "[-h help (this message)]" << std::endl << std::endl;
			
		if(num_args > 2) std::cerr << "Error: Too many arguments." << std::endl;
		if(num_args < 2) std::cerr << "Error: No arguments given." << std::endl;

		return 0;
		
	}
	
	std::istringstream ss(argv[optind]);
	int file_size;
	if (!(ss >> file_size)) {
		std::cerr << "File size is not a valid number (" << argv[optind] << ") Please enter a valid integer for file size." << std::endl;
		return 0;
	} else if (!ss.eof()) {
		std::cerr << "Trailing characters found after file size (" << argv[optind] << ") Please enter a valid integer for file size." << std::endl;
		return 0;
	}

	char *fastq_directory = argv[optind + 1];
	struct stat check_dir;
	
	if(stat (fastq_directory, &check_dir) != 0){
		std::cerr << "Directory path given (" << fastq_directory << ") is not a valid path. Please enter a valid directory." << std::endl;
		return 0;
	}
	
	std::cerr << "file size: " << file_size << std::endl;
	std::cerr << "fastq_directory: " << fastq_directory << std::endl;
	
	// Create a new connection
	Connection new_con(host, port, verbose);
	std::cerr << "Client connection established" << std::endl;
	
	std::unique_ptr<DeviceService::Stub> dev_stub_ =  DeviceService::NewStub(new_con.get_channel());
	std::unique_ptr<AcquisitionService::Stub> acq_stub_ = AcquisitionService::NewStub(new_con.get_channel());
	std::unique_ptr<ProtocolService::Stub> proto_stub_ = ProtocolService::NewStub(new_con.get_channel());
	
	std::cerr << "Checking for successful connection by getting the number of channels from the MinION." << std::endl;
	ClientContext cell_context;
	::minknow_api::device::GetFlowCellInfoRequest cell_request;
	::minknow_api::device::GetFlowCellInfoResponse cell_response;
	::grpc::Status return_status = dev_stub_->get_flow_cell_info(&cell_context, cell_request, &cell_response);
	
	if(cell_response.channel_count() == 0){
		std::cerr << "Unable to establish connection to MinKNOW. Please double check that you have the right host and port and try again. Exiting." << std::endl;
		return 0;
	}
	
	std::cerr << "Max number of channels: " << cell_response.channel_count() << std::endl;
	
	std::string fastq_directory_str(fastq_directory);
	
	int total_data_read = 0;
	
	while(true){
		std::cerr << "Checking directory..." << std::endl;
		total_data_read = 0;
		struct dirent *entry;
		DIR *dir = opendir(fastq_directory);
		while (entry = readdir(dir)) {
			std::string filename = entry->d_name;
			if(filename.substr(filename.find_last_of(".") + 1) == "fastq") {
				std::string file_path = fastq_directory_str + "\\" + filename;
				std::cerr << file_path << std::endl;
				std::ifstream f(file_path.c_str());
				if(!f.is_open()){
					std::cerr << "Error opening file." << std::endl;
					break;
				}
				std::string line;
				int line_num = 1;
				while(!f.eof()){
					getline(f,line);
					if(line_num % 2 == 0 && line_num % 4 != 0){
						total_data_read += line.length();
					}
					line_num++;
				}
				f.close();
			} 
		}
		closedir(dir);
		if(total_data_read >= file_size){
			break;
		}
		std::cerr << "Not enough data found. Waiting for " << wait_time << " seconds..." << std::endl;
		sleep(wait_time);
	}

	std::cerr << "Stopping MinKNOW run as we have read in all the data we wanted to:" << std::endl;
	// ClientContext stop_context;
	// ::minknow_api::acquisition::StopRequest stop_request;
	// // ::minknow_api::acquisition::StopRequest_DataAction action = StopRequest_DataAction_STOP_DEFAULT;
	// ::minknow_api::acquisition::StopRequest_DataAction action = StopRequest_DataAction_STOP_KEEP_ALL_DATA;
	// // ::minknow_api::acquisition::StopRequest_DataAction action = StopRequest_DataAction_STOP_FINISH_PROCESSING;
	// stop_request.set_data_action_on_stop(action);
	
	// ::minknow_api::acquisition::StopResponse stop_response;
	// ::grpc::Status status1 = acq_stub_->stop(&stop_context, stop_request, &stop_response);
	ClientContext stop_context;
	::minknow_api::protocol::StopProtocolRequest stop_request;
	::minknow_api::acquisition::StopRequest_DataAction action = StopRequest_DataAction_STOP_KEEP_ALL_DATA;
	stop_request.set_data_action_on_stop(action);
	::minknow_api::protocol::StopProtocolResponse stop_response;
	::grpc::Status status_stop = proto_stub_->stop_protocol(&stop_context, stop_request, &stop_response);
	
	
	return 0;
}
